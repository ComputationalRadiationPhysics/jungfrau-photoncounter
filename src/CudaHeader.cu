#include "CudaHeader.hpp"
#include <stdio.h>

void handleCudaError(hipError_t error, const char* file, int line) {
    if(error == hipSuccess) return;
    printf("<%s>:%i",file,line);
    printf(" %s\n", hipGetErrorString(error));
}
