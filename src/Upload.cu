#include "Upload.hpp"

std::size_t Uploader::nextFree = 0;
std::size_t Uploader::dimX = 0;
std::size_t Uploader::dimY = 0;
std::vector<deviceData> Uploader::devices;

bool isMapEmpty(Datamap map, std::size_t dimX, std::size_t dimY) {
	for(std::size_t y = 0; y < dimY; ++y) {
		for(std::size_t x = 0; x < dimX; ++x) {
			if(map(x, y) != 0)
				return false;
		}
	}
	return true;
}

void handleCudaError(hipError_t error, const char* file, int line)
{
    if (error != hipSuccess) {
        char errorString[1000];
        snprintf(errorString, 1000,
                 "FATAL ERROR (CUDA, %d): %s in %s at line %d!\n", error,
                 hipGetErrorString(error), file, line);
        fputs(errorString, stderr);
        exit(EXIT_FAILURE);
    }
}

Uploader::Uploader(std::array<Gainmap, 3> gain,
                   std::array<Pedestalmap, 3> pedestal, std::size_t dimX,
                   std::size_t dimY, std::size_t numberOfDevices)
    : gain(gain), pedestal(pedestal),/* dimX(dimX), dimY(dimY),*/
      resources(STREAMS_PER_GPU * numberOfDevices)
 {
	 Uploader::dimX = dimX;
	 Uploader::dimY = dimY;
	 DEBUG("Entering uploader constructor!");
	 printDeviceName();
	 devices.resize(resources.getSize());

	 DEBUG("Initializing GPUs!");
	 initGPUs();
	 // TODO: init pedestal maps
	 currentBlock.reserve(GPU_FRAMES);
	 DEBUG("elements in the ringbuffer: " << resources.getNumberOfElements());
	 DEBUG("End of constructor!");
 }

 Uploader::~Uploader() { freeGPUs(); }

void Uploader::printDeviceName() {
	struct hipDeviceProp_t prop;
	int numDevices;

	HANDLE_CUDA_ERROR(hipGetDeviceCount(&numDevices));
	for(int i = 0; i < numDevices; ++i) {
		HANDLE_CUDA_ERROR(hipSetDevice(i));
		HANDLE_CUDA_ERROR(hipGetDeviceProperties(&prop, i));
		std::cout << "Device #" << i << ":\t" << prop.name << std::endl;
	}
}

bool Uploader::upload(std::vector<Datamap>& data)
{
	//TODO: waht to do with a small amount of frames when terminating?
	DEBUG("uploading " << data.size() << "maps");
	for (std::size_t i = 0; i < data.size(); ++i) {
		if (currentBlock.size() == GPU_FRAMES) {
			if (!calcFrames(currentBlock)) {
				//TODO: find a better solution below
				//remove all used frames from the front
				for(std::size_t j = data.size() - i; j > 0; --j) {
					data[j-1] = data[i+j-1];
				}

				for(std::size_t j = 0; j < i; ++j)
					data.pop_back();

				DEBUG("new size at " << i << " = " << data.size());
				return false;
			}

			currentBlock.clear();
		}
		currentBlock.push_back(data[i]);
	}

	DEBUG("getting out! Resources available: " << resources.getNumberOfElements());
	data.clear();
	return true;
}

 std::vector<Photonmap> Uploader::download()
 {
	 std::vector<Photonmap> ret;
	 int current = nextFree;

	 if(devices[nextFree].state != READY)
		 return ret;
	 nextFree = (nextFree + 1) % resources.getSize();



	 std::size_t numPhotons = dimX * dimY * GPU_FRAMES;
	 struct deviceData* dev = &Uploader::devices[current];

	 for (size_t i = 0; i < numPhotons; i += dimX * dimY) {
		 //TODO: use emplace back directly with ret
		 dev->photon_host.emplace_back(dimX, dimY, &dev->photon_host_raw[i]);
	 }

	 //TODO: remove debug below
	 /*for(std::size_t o = 0; o < GPU_FRAMES; ++o){
		 if(isMapEmpty(dev->photon_host[o], dimX, dimY))
			 DEBUG("map " << o << " is empty!");
			 }*/



	 ret = Uploader::devices[current].photon_host;
	 Uploader::devices[current].photon_host.clear();
	 DEBUG("setting " << current << " to FREE");
	 Uploader::devices[current].state = FREE;
	 if(!resources.push(&devices[current])) {
		 fputs("FATAL ERROR (RingBuffer): Unexpected size!\n", stderr);
		 exit(EXIT_FAILURE);
	 }
	 DEBUG("resources in use: " << resources.getNumberOfElements());
	 return ret;
 }

 void Uploader::initGPUs()
 {
	 DEBUG("initGPU()");

	 //TODO: init pedestalmaps!
	 for (std::size_t i = 0; i < devices.size(); ++i) {
		 DEBUG("Uploading Pedestalmaps for device " << i / STREAMS_PER_GPU << " with i=" << i);
		 devices[i].gain_host = &gain;

		 DEBUG("Uploading Gainmaps for device " << i / STREAMS_PER_GPU << " with i=" << i);
		 devices[i].pedestal_host = &pedestal;

		 DEBUG("setting " << i << " to FREE");
		 devices[i].state = FREE;
		 //TODO: is this really needed? if yes, throw out device member
		 devices[i].id = i;
		 devices[i].device = i / STREAMS_PER_GPU;

		 DEBUG("Setting device " << i / STREAMS_PER_GPU);
		 HANDLE_CUDA_ERROR(hipSetDevice(i / STREAMS_PER_GPU));

		 DEBUG("Allocating GPU memory on device for #" << i);
		 HANDLE_CUDA_ERROR(hipMalloc((void**)&devices[i].gain, dimX * dimY * sizeof(double) * 3));
		 HANDLE_CUDA_ERROR(hipMalloc((void**)&devices[i].pedestal, dimX * dimY * sizeof(uint16_t) * 3));
		 HANDLE_CUDA_ERROR(hipMalloc((void**)&devices[i].data, dimX * dimY * sizeof(uint16_t) * GPU_FRAMES));
		 HANDLE_CUDA_ERROR(hipMalloc((void**)&devices[i].photons, dimX * dimY * sizeof(uint16_t) * GPU_FRAMES));

		 HANDLE_CUDA_ERROR(hipHostMalloc((void**)&devices[i].data_pinned, dimX * dimY * sizeof(uint16_t) * GPU_FRAMES));
		 HANDLE_CUDA_ERROR(hipHostMalloc((void**)&devices[i].photon_pinned, dimX * dimY * sizeof(uint16_t) * GPU_FRAMES));

		 DEBUG("Creating GPU stream #" << i);
		 HANDLE_CUDA_ERROR(hipStreamCreate(&devices[i].str));

		 synchronize();

		 DEBUG("Uploading Gainmaps for #" << i);
		 uploadGainmap(devices[i]);
		 DEBUG("Uploading Pedestalmaps for #" << i);
		 uploadPedestalmap(devices[i]);

		 DEBUG("elements in the ringbuffer: " << resources.getNumberOfElements());
		 DEBUG("is rb empty? " << resources.isEmpty());
		 DEBUG("is rb full? " << resources.isFull());

		 if (!resources.push(&devices[i])) {
			 fputs("FATAL ERROR (RingBuffer): Unexpected size!\n", stderr);
			 exit(EXIT_FAILURE);
		 }
	 }
	 DEBUG("number of elements in resources: " << resources.getNumberOfElements());
	 DEBUG("initGPU done!");
 }

 void Uploader::freeGPUs()
 {
	 synchronize();
	 for (std::size_t i = 0; i < devices.size(); ++i) {
		 HANDLE_CUDA_ERROR(hipSetDevice(devices[i].device));
		 HANDLE_CUDA_ERROR(hipFree(devices[i].gain));
		 HANDLE_CUDA_ERROR(hipFree(devices[i].pedestal));
		 HANDLE_CUDA_ERROR(hipFree(devices[i].data));
		 HANDLE_CUDA_ERROR(hipFree(devices[i].photons));
		 HANDLE_CUDA_ERROR(hipStreamDestroy(devices[i].str));
	 }
 }

 void Uploader::synchronize()
 {
	 for (struct deviceData dev : devices)
		 HANDLE_CUDA_ERROR(hipStreamSynchronize(dev.str));
 }

 void Uploader::uploadGainmap(struct deviceData stream)
 {
	 DEBUG("Gainmap upload ...");
	 HANDLE_CUDA_ERROR(hipSetDevice(stream.device));
	 DEBUG("hipMemcpy(" << stream.gain << ", " << stream.gain_host->at(0).data() << ", " << stream.gain_host->at(0).getSizeBytes() * 3 << ", hipMemcpyHostToDevice);");
	 HANDLE_CUDA_ERROR(hipMemcpy(stream.gain, stream.gain_host->at(0).data(), stream.gain_host->at(0).getSizeBytes() * 3, hipMemcpyHostToDevice));
	 DEBUG("Done!");
 }

 void Uploader::uploadPedestalmap(struct deviceData stream)
 {
	 DEBUG("Pedestalmap upload ...");
	 HANDLE_CUDA_ERROR(hipSetDevice(stream.device));
	 HANDLE_CUDA_ERROR(hipMemcpy(stream.pedestal, stream.pedestal_host->at(0).data(), stream.pedestal_host->at(0).getSizeBytes() * 3, hipMemcpyHostToDevice));
	 DEBUG("Done!");
 }

 void Uploader::downloadGainmap(struct deviceData stream)
 {
	 DEBUG("Gainmap upload ...");
	 HANDLE_CUDA_ERROR(hipSetDevice(stream.device));
	 DEBUG("hipMemcpy(" << stream.gain_host->at(0).data() << ", " << stream.gain << ", " << stream.gain_host->at(0).getSizeBytes() * 3 << ", hipMemcpyHostToDevice);");
	 HANDLE_CUDA_ERROR(hipMemcpy(stream.gain_host->at(0).data(), stream.gain, stream.gain_host->at(0).getSizeBytes() * 3, hipMemcpyDeviceToHost));
	 DEBUG("Done!");
 }

 void Uploader::downloadPedestalmap(struct deviceData stream)
 {
	 DEBUG("Pedestalmap doanload ...");
	 HANDLE_CUDA_ERROR(hipSetDevice(stream.device));
	 HANDLE_CUDA_ERROR(hipMemcpy(stream.pedestal_host->at(0).data(), stream.pedestal, stream.pedestal_host->at(0).getSizeBytes() * 3, hipMemcpyDeviceToHost));
	 DEBUG("Done!");
 }

 bool Uploader::calcFrames(std::vector<Datamap>& data)
 {
	 //DEBUG("calcFrames");
	 std::vector<Photonmap> photonMaps;
	 photonMaps.reserve(GPU_FRAMES);

	 if(data.empty()) {
		 DEBUG("no data .... doing nothing");
		 return false;
	 }

	 struct deviceData* dev;
	 if(!resources.pop(dev))
		 return false;


    std::size_t numPhotons = dimX * dimY * GPU_FRAMES;
    dev->photon_host_raw = (uint16_t*)malloc(numPhotons * sizeof(uint16_t));
    if (!dev->photon_host_raw) {
        fputs("FATAL ERROR (Memory): Allocation failed!\n", stderr);
        exit(EXIT_FAILURE);
    }

	 DEBUG("copyin to pinned memory");
	 DEBUG("pinned data = " << dev->data_pinned << " & src = " << data[0].data());
	 HANDLE_CUDA_ERROR(hipMemcpyAsync(dev->data_pinned, data[0].data(), dimX * dimY * GPU_FRAMES, hipMemcpyHostToHost, dev->str));

	 DEBUG("Doing GPU stuff now");

	 DEBUG("setting " << dev->id << " to PROCESSING");
	 dev->state = PROCESSING;
	 uploadToGPU(*dev, data);

	 calculate<<<dimX, dimY, 3 * (sizeof(uint16_t) + sizeof(double)) * dimY, dev->str>>>(uint16_t(dimX * dimY), dev->pedestal, dev->gain, dev->data, uint16_t(GPU_FRAMES), dev->photons);
     CHECK_CUDA_KERNEL;

	 downloadFromGPU(*dev);
	 
	 DEBUG("copying data from gpu to pinned memory");
	 HANDLE_CUDA_ERROR(hipMemcpyAsync(dev->photon_host_raw, dev->photon_pinned, dimX * dimY * GPU_FRAMES, hipMemcpyHostToHost, dev->str));

	 DEBUG("Creating callback ...");
	 HANDLE_CUDA_ERROR(hipStreamAddCallback(dev->str, Uploader::callback, &dev->id, 0));

	 return true;
 }

 void CUDART_CB Uploader::callback(hipStream_t stream, hipError_t status, void* data) {
	 //suppress "unused variable" compiler warning
	 (void)stream;

	 DEBUG("HELP ME I AM TRAPPED IN A SUPERCOMPUTER AND I CAN'T GET OUT!!!!");

	 if(data == NULL) {
		 fputs("FATAL ERROR (callback): Missing index!\n", stderr);
		 exit(EXIT_FAILURE);
	 }

	 HANDLE_CUDA_ERROR(status);
	 DEBUG("setting " << *((int*)data) << " to READY");

	 struct deviceData* dev = &Uploader::devices[*((int*)data)];
	 dev->state = READY;
	 DEBUG("stream: " << *((int*)data));
 }

void Uploader::uploadToGPU(struct deviceData& dev, std::vector<Datamap>& data)
{
	if(data.empty())
		return;
    HANDLE_CUDA_ERROR(hipSetDevice(dev.device));
	DEBUG("upload size: " << data.size() * data[0].getSizeBytes());

	//TODO: clean up
	HANDLE_CUDA_ERROR(hipMemcpyAsync(dev.data, dev.data_pinned, data.size() * data[0].getSizeBytes(), hipMemcpyHostToDevice, dev.str));
}

void Uploader::downloadFromGPU(struct deviceData& dev)
{
    std::size_t numPhotons = dimX * dimY * GPU_FRAMES;
	std::size_t copySize = numPhotons * sizeof(*dev.photons);

    HANDLE_CUDA_ERROR(hipSetDevice(dev.device));
	HANDLE_CUDA_ERROR(hipMemcpyAsync(dev.photon_pinned, dev.photons, copySize, hipMemcpyDeviceToHost, dev.str));
}

