#include "upload.hpp"

static void handleCudaError(hipError_t error, const char* file, int line) {
	if(err != hipSuccess) {
		char errorString[1000];
		snprintf(errorString, 1000, "%s in %s at line %d!\n", hipGetErrorString(error), file, line);
		perror(errorString);
		exit(EXIT_FAILURE);
	}
}

Uploader::Uploader(Gainmap& gain, Pedestalmap& pedastel) : gain(gain), pedestal(pedestal), input_buffer(RINGBUFFER_SIZE), output_buffer(RINGBUFFER_SIZE), mem_usage(0){
	HANDLE_CUDA_ERROR(hipMalloc((void**)&gain_device, gain->getSizeBytes()));
	HANDLE_CUDA_ERROR(hipMalloc((void**)&pedestal_device, pedestal->getSizeBytes()));
	//TODO: find a better solution than this ugly hack below (using pedestal->getSizeBytes())!
	HANDLE_CUDA_ERROR(hipMalloc((void**)&data_device, pedestal->getSizeBytes()));
	uploadGainmap(*gain);
	uploadPedestalmap(*pedestal);
	current_block.reserve(GPU_FRAMES);
}

Uploader::~Uploader() {
	HANDLE_CUDA_ERROR(hipFree(gain_device));
	HANDLE_CUDA_ERROR(hipFree(pedestal_device));
}

void Uploader::upload(std::vector<Datamap> data) {
	size_t i = 0;
	while(i < data.size()) {
		current_block.push_back(data[i]);
		if(current_block.size() == GPU_FRAMES) {
			input_buffer.push(current_block);
			current_block.clear();
		}
	}
}

std::vector<Datamap> Uploader::download() {
	std::vector<Datamap> ret;
	output_buffer.pop(ret);
	return ret;
}

void Uploader::uploadGainmap() {
	HANDLE_CUDA_ERROR(hipMemcpy(gain_device, gain->data(), gain->getSizeBytes(), hipMemcpyHostToDevice));
}

void Uploader::uploadPedestalmap() {
	HANDLE_CUDA_ERROR(hipMemcpy(pedestal_device, pedestal->data(), pedestal->getSizeBytes(), hipMemcpyHostToDevice));
}

void Uploader::downloadGainmap() {
	HANDLE_CUDA_ERROR(hipMemcpy(gain->data(), gain_device, gain->getSizeBytes(), hipMemcpyDeviceToHost));
}

void Uploader::downloadPedestalmap() {
	HANDLE_CUDA_ERROR(hipMemcpy(pedestal->data(), pedestal_device, pedestal->getSizeBytes(), hipMemcpyDeviceToHost));
}

void Uploader::uploadToGPU() {
	std::vector<Datamap> to_upload;
	input_buffer.pop(to_upload);
	//TODO: better error handling!
	if(to_upload.empty())
		return;
	//TODO: figure out how to multithread this properly!

}

void Uploader::downloadFromGPU() {
	//TODO: figure out how to multithread this properly!
}
