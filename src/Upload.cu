#include "Upload.hpp"

std::size_t Uploader::nextFree = 0;
std::vector<deviceData> Uploader::devices;

void handleCudaError(hipError_t error, const char* file, int line)
{
    if (error != hipSuccess) {
        char errorString[1000];
        snprintf(errorString, 1000,
                 "FATAL ERROR (CUDA, %d): %s in %s at line %d!\n", error,
                 hipGetErrorString(error), file, line);
        fputs(errorString, stderr);
        exit(EXIT_FAILURE);
    }
}

Uploader::Uploader(std::array<Gainmap, 3> gain,
                   std::array<Pedestalmap, 3> pedestal, std::size_t dimX,
                   std::size_t dimY, std::size_t numberOfDevices)
    : gain(gain), pedestal(pedestal), dimX(dimX), dimY(dimY),
      resources(2 * numberOfDevices)
 {
	 DEBUG("Entering uploader constructor!");
	 printDeviceName();
	 devices.resize(resources.getSize());

	 DEBUG("Initializing GPUs!");
	 initGPUs();
	 // TODO: init pedestal maps
	 DEBUG("Reserving memory for currentBlock.");
	 currentBlock.reserve(GPU_FRAMES);
	 DEBUG("elements in the ringbuffer: " << resources.getNumberOfElements());
	 DEBUG("is rb empty? " << resources.isEmpty());
	 DEBUG("is rb full? " << resources.isFull());
	 DEBUG("End of constructor!");
 }

 Uploader::~Uploader() { freeGPUs(); }

void Uploader::printDeviceName() {
	struct hipDeviceProp_t prop;
	int numDevices;

	HANDLE_CUDA_ERROR(hipGetDeviceCount(&numDevices));
	for(int i = 0; i < numDevices; ++i) {
		HANDLE_CUDA_ERROR(hipSetDevice(i));
		HANDLE_CUDA_ERROR(hipGetDeviceProperties(&prop, i));
		std::cout << "Device #" << i << ":\t" << prop.name << std::endl;
	}
}

bool Uploader::upload(std::vector<Datamap>& data)
{
	//DEBUG("Entering upload");
	//TODO: handle incomplete uploads
	std::size_t i = 0;
	while (i < data.size()) {
		currentBlock.push_back(data[i++]);
		if (currentBlock.size() == GPU_FRAMES) {
			//			DEBUG("preparing upload to gpu");
			if (!calcFrames(currentBlock)) {
				//DEBUG("rearranging data...");
				//TODO: find a better solution below
				//remove all used frames from the front
				for(std::size_t j = data.size() - i; j > 0; --j) {
					data[j-1] = data[i+j-1];
					data.pop_back();
				}
				//DEBUG("done");
				currentBlock.clear();
				return false;
			}
			currentBlock.clear();
		}
		else if(currentBlock.size() > GPU_FRAMES) {
			DEBUG("This was never meant to happen ...");
			DEBUG("Commiting suicide ...");
			exit(-1);
		}
	}
	return true;
}

 std::vector<Photonmap> Uploader::download()
 {
	 std::vector<Photonmap> ret;
	 int current = nextFree;
	 if(devices[nextFree].state != READY)
		 return ret;
	 ++nextFree;

	 ret = Uploader::devices[current].photon_host;
	 Uploader::devices[current].photon_host.clear();
	 if(!resources.push(&devices[current])) {
		 fputs("FATAL ERROR (RingBuffer): Unexpected size!\n", stderr);
		 exit(EXIT_FAILURE);
	 }
	 return ret;
 }

 void CUDART_CB Uploader::callback(hipStream_t stream, hipError_t status, void* data) {
	 //suppress "unused variable" compiler warning
	 (void)stream;

	 DEBUG("HELP ME I AM TRAPPED IN A SUPERCOMPUTER AND I CAN'T GET OUT!!!!");

	 if(data == NULL) {
		 fputs("FATAL ERROR (callback): Missing index!\n", stderr);
		 exit(EXIT_FAILURE);
	 }

	 HANDLE_CUDA_ERROR(status);
	 Uploader::devices[*((int*)data)].state = READY;
	 DEBUG("stream: " << *((int*)data));
 }

 void Uploader::initGPUs()
 {
	 DEBUG("initGPU()");

	 //TODO: init pedestalmaps!
	 for (std::size_t i = 0; i < devices.size(); ++i) {
		 DEBUG("Uploading Pedestalmaps for device " << i / 2 << " with i=" << i);
		 devices[i].gain_host = &gain;

		 DEBUG("Uploading Gainmaps for device " << i / 2 << " with i=" << i);
		 devices[i].pedestal_host = &pedestal;

		 devices[i].state = FREE;
		 //TODO: is this really needed? if yes, throw out device member
		 devices[i].id = i;
		 devices[i].device = i / 2;

		 DEBUG("Setting device " << i / 2);
		 HANDLE_CUDA_ERROR(hipSetDevice(i / 2));

		 DEBUG("Allocating GPU memory on device for #" << i);
		 HANDLE_CUDA_ERROR(hipMalloc((void**)&devices[i].gain, dimX * dimY * sizeof(double) * 3));
		 HANDLE_CUDA_ERROR(hipMalloc((void**)&devices[i].pedestal, dimX * dimY * sizeof(uint16_t) * 3));
		 HANDLE_CUDA_ERROR(hipMalloc((void**)&devices[i].data, dimX * dimY * sizeof(uint16_t) * GPU_FRAMES));
		 HANDLE_CUDA_ERROR(hipMalloc((void**)&devices[i].photons, dimX * dimY * sizeof(uint16_t) * GPU_FRAMES));

		 DEBUG("Creating GPU stream #" << i);
		 HANDLE_CUDA_ERROR(hipStreamCreate(&devices[i].str));

		 synchronize();

		 DEBUG("Uploading Gainmaps for #" << i);
		 uploadGainmap(devices[i]);
		 DEBUG("Uploading Pedestalmaps for #" << i);
		 uploadPedestalmap(devices[i]);

		 DEBUG("elements in the ringbuffer: " << resources.getNumberOfElements());
		 DEBUG("is rb empty? " << resources.isEmpty());
		 DEBUG("is rb full? " << resources.isFull());

		 if (!resources.push(&devices[i])) {
			 fputs("FATAL ERROR (RingBuffer): Unexpected size!\n", stderr);
			 exit(EXIT_FAILURE);
		 }
	 }
	 DEBUG("number of elements in resources: " << resources.getNumberOfElements());
	 DEBUG("initGPU done!");
 }

 void Uploader::freeGPUs()
 {
	 synchronize();
	 for (std::size_t i = 0; i < devices.size(); ++i) {
		 HANDLE_CUDA_ERROR(hipSetDevice(devices[i].device));
		 HANDLE_CUDA_ERROR(hipFree(devices[i].gain));
		 HANDLE_CUDA_ERROR(hipFree(devices[i].pedestal));
		 HANDLE_CUDA_ERROR(hipFree(devices[i].data));
		 HANDLE_CUDA_ERROR(hipFree(devices[i].photons));
		 HANDLE_CUDA_ERROR(hipStreamDestroy(devices[i].str));
	 }
 }

 void Uploader::synchronize()
 {
	 for (struct deviceData dev : devices)
		 HANDLE_CUDA_ERROR(hipStreamSynchronize(dev.str));
 }

 void Uploader::uploadGainmap(struct deviceData stream)
 {
	 DEBUG("Gainmap upload ...");
	 HANDLE_CUDA_ERROR(hipSetDevice(stream.device));
	 DEBUG("hipMemcpy(" << stream.gain << ", " << stream.gain_host->at(0).data() << ", " << stream.gain_host->at(0).getSizeBytes() * 3 << ", hipMemcpyHostToDevice);");
	 HANDLE_CUDA_ERROR(hipMemcpy(stream.gain, stream.gain_host->at(0).data(), stream.gain_host->at(0).getSizeBytes() * 3, hipMemcpyHostToDevice));
	 DEBUG("Done!");
 }

 void Uploader::uploadPedestalmap(struct deviceData stream)
 {
	 DEBUG("Pedestalmap upload ...");
	 HANDLE_CUDA_ERROR(hipSetDevice(stream.device));
	 HANDLE_CUDA_ERROR(hipMemcpy(stream.pedestal, stream.pedestal_host->at(0).data(), stream.pedestal_host->at(0).getSizeBytes() * 3, hipMemcpyHostToDevice));
	 DEBUG("Done!");
 }

 void Uploader::downloadGainmap(struct deviceData stream)
 {
	 DEBUG("Gainmap upload ...");
	 HANDLE_CUDA_ERROR(hipSetDevice(stream.device));
	 DEBUG("hipMemcpy(" << stream.gain_host->at(0).data() << ", " << stream.gain << ", " << stream.gain_host->at(0).getSizeBytes() * 3 << ", hipMemcpyHostToDevice);");
	 HANDLE_CUDA_ERROR(hipMemcpy(stream.gain_host->at(0).data(), stream.gain, stream.gain_host->at(0).getSizeBytes() * 3, hipMemcpyDeviceToHost));
	 DEBUG("Done!");
 }

 void Uploader::downloadPedestalmap(struct deviceData stream)
 {
	 DEBUG("Pedestalmap doanload ...");
	 HANDLE_CUDA_ERROR(hipSetDevice(stream.device));
	 HANDLE_CUDA_ERROR(hipMemcpy(stream.pedestal_host->at(0).data(), stream.pedestal, stream.pedestal_host->at(0).getSizeBytes() * 3, hipMemcpyDeviceToHost));
	 DEBUG("Done!");
 }

 bool Uploader::calcFrames(std::vector<Datamap>& data)
 {
	 //DEBUG("calcFrames");
	 std::vector<Photonmap> photonMaps;
	 photonMaps.reserve(GPU_FRAMES);

	 if(data.empty()) {
		 DEBUG("no data .... doing nothing");
		 return false;
	 }

	 struct deviceData* dev;
	 if(!resources.pop(dev))
		 return false;

	 DEBUG("Doing GPU stuff now");

	 uploadToGPU(*dev, data);

	 calculate<<<dimX, dimY, 3 * (sizeof(uint16_t) + sizeof(double)) * dimY, dev->str>>>(uint16_t(dimX * dimY), dev->pedestal, dev->gain, dev->data, uint16_t(GPU_FRAMES), dev->photons);
     CHECK_CUDA_KERNEL;
	 downloadFromGPU(*dev);

	 DEBUG("Creating callback ...");
	 HANDLE_CUDA_ERROR(hipStreamAddCallback(dev->str, Uploader::callback, &dev->id, 0));

	 //DEBUG("End calcFrames");
	 return true;
 }

void Uploader::uploadToGPU(struct deviceData& dev, std::vector<Datamap>& data)
{
    HANDLE_CUDA_ERROR(hipSetDevice(dev.device));
	//TODO: is data.data() the right thing?
	//TODO: used pinned memory?
    HANDLE_CUDA_ERROR(hipMemcpyAsync(dev.data, data.data(), data.size() * sizeof(data[0]), hipMemcpyHostToDevice, dev.str));
}

void Uploader::downloadFromGPU(struct deviceData& dev)
{
    //DEBUG("Entering downloadFromGPU (str=" << dev.str << ")");
    std::size_t numPhotons = dimX * dimY * GPU_FRAMES;
    //DEBUG("numPhotons = " << numPhotons);
    uint16_t* photonData = (uint16_t*)malloc(numPhotons * sizeof(uint16_t));
    if (!photonData) {
        fputs("FATAL ERROR (Memory): Allocation failed!\n", stderr);
        exit(EXIT_FAILURE);
    }
    DEBUG(numPhotons * sizeof(uint16_t) << " Bytes allocated");
    //DEBUG("hipMemcpyAsync(" << photonData << ", " << dev.photons << ", " << numPhotons * sizeof(uint16_t) << ", hipMemcpyDeviceToHost, " << dev.str << ");");

    HANDLE_CUDA_ERROR(hipSetDevice(dev.device));
    HANDLE_CUDA_ERROR(hipMemcpyAsync(photonData, dev.photons, numPhotons * sizeof(uint16_t), hipMemcpyDeviceToHost, dev.str));

    //DEBUG("data downloaded");

    for (size_t i = 0; i < numPhotons; i += dimX * dimY) {
        dev.photon_host.emplace_back(dimX, dimY, &photonData[i]);
    }
    //DEBUG("Data written. downloadFromGPU done!");
}

