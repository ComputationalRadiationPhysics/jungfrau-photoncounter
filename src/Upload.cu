#include "Upload.hpp"

std::size_t Uploader::nextFree = 0;
std::vector<deviceData> devices;

static void handleCudaError(hipError_t error, const char* file, int line)
{
    if (error != hipSuccess) {
        char errorString[1000];
        snprintf(errorString, 1000,
                 "FATAL ERROR (CUDA, %d): %s in %s at line %d!\n", error,
                 hipGetErrorString(error), file, line);
        fputs(errorString, stderr);
        exit(EXIT_FAILURE);
    }
}

Uploader::Uploader(std::array<Gainmap, 3> gain,
                   std::array<Pedestalmap, 3> pedestal, std::size_t dimX,
                   std::size_t dimY, std::size_t numberOfDevices)
    : gain(gain), pedestal(pedestal), dimX(dimX), dimY(dimY),
      resources(/*2 * numberOfDevices*/1)
 {
	 DEBUG("Entering uploader constructor!");
	 // TODO: FIX MULTI GPU
	 DEBUG("FIXME: Forcing to use only one GPU (with one stream)!");
	 //TODO: use size function of ringbuffer (implement maybe)
	 devices.resize(2 * numberOfDevices);

	 DEBUG("Initializing GPUs!");
	 initGPUs();
	 // TODO: init pedestal maps
	 DEBUG("Reserving memory for currentBlock.");
	 currentBlock.reserve(GPU_FRAMES);
	 DEBUG("End of constructor!");
 }

 Uploader::~Uploader() { freeGPUs(); }

 bool Uploader::upload(std::vector<Datamap> data)
 {
	 std::size_t i = 0;
	 while (i < data.size()) {
		 currentBlock.push_back(data[i]);
		 if (currentBlock.size() == GPU_FRAMES) {
			 // input_buffer.push(current_block);
			 if (!calcFrames(currentBlock))
				 return false;
			 currentBlock.clear();
		 }
	 }
	 return true;
 }

 std::vector<Photonmap> Uploader::download()
 {
	 std::vector<Photonmap> ret;
	 int current = nextFree;
	 if(devices[nextFree].state != READY)
		 return ret;
	 ++nextFree;

	 /*	//TODO: use local photonmap 
	 for(size_t i = 0; i < GPU_FRAMES * dimX * dimY; ++i){
			 devices[current].photon_host.emplace_back(dimX, dimY, devices[current].photon);
			 }*/

	 ret = Uploader::devices[current].photon_host;
	 Uploader::devices[current].photon_host.clear();
	 if(!resources.push(&devices[current])) {
		 fputs("FATAL ERROR (RingBuffer): Unexpected size!", stderr);
		 exit(EXIT_FAILURE);
	 }
	 return ret;
 }

 void Uploader::callback(void* data) {
	 //TODO: does this work like that
	 Uploader::devices[*((int*)data)].state = READY;
 }

 void Uploader::initGPUs()
 {
	 DEBUG("initGPU()");

	 for (std::size_t i = 0; i < devices.size(); ++i) {
		 DEBUG("Uploading Pedestalmaps for device " << i / 2 << " with i=" << i);
		 devices[i].gain_host = &gain;

		 DEBUG("Uploading Gainmaps for device " << i / 2 << " with i=" << i);
		 devices[i].pedestal_host = &pedestal;

		 devices[i].state = FREE;
		 //TODO: is this really needed? if yes, throw out device member
		 devices[i].id = i;

		 DEBUG("Setting device " << i / 2);
		 HANDLE_CUDA_ERROR(hipSetDevice(i / 2));

		 // TODO: fix dimensions!!!
		 DEBUG("Allocating GPU memory on device for #" << i);
		 HANDLE_CUDA_ERROR(hipMalloc((void**)&devices[i].gain, dimX * dimY * sizeof(double) * 3));
		 HANDLE_CUDA_ERROR(hipMalloc((void**)&devices[i].pedestal, dimX * dimY * sizeof(uint16_t) * 3));
		 HANDLE_CUDA_ERROR(hipMalloc((void**)&devices[i].data, dimX * dimY * sizeof(uint16_t) * GPU_FRAMES));
		 HANDLE_CUDA_ERROR(hipMalloc((void**)&devices[i].photons, dimX * dimY * sizeof(uint16_t) * GPU_FRAMES));

		 DEBUG("Creating GPU stream #" << i);
		 HANDLE_CUDA_ERROR(hipStreamCreate(&devices[i].str));

		 synchronize();

		 DEBUG("Uploading Gainmaps for #" << i);
		 uploadGainmap(devices[i]);
		 DEBUG("Uploading Pedestalmaps for #" << i);
		 uploadPedestalmap(devices[i]);

		 if (!resources.push(&devices[i])) {
			 fputs("FATAL ERROR (RingBuffer): Unexpected size!", stderr);
			 exit(EXIT_FAILURE);
		 }
	 }
	 DEBUG("initGPU done!");
 }

 void Uploader::freeGPUs()
 {
	 synchronize();
	 for (std::size_t i = 0; i < devices.size(); ++i) {
		 HANDLE_CUDA_ERROR(hipSetDevice(devices[i].device));
		 HANDLE_CUDA_ERROR(hipFree(devices[i].gain));
		 HANDLE_CUDA_ERROR(hipFree(devices[i].pedestal));
		 HANDLE_CUDA_ERROR(hipFree(devices[i].data));
		 HANDLE_CUDA_ERROR(hipFree(devices[i].photons));
		 HANDLE_CUDA_ERROR(hipStreamDestroy(devices[i].str));
	 }
 }

 void Uploader::synchronize()
 {
	 for (struct deviceData dev : devices)
		 HANDLE_CUDA_ERROR(hipStreamSynchronize(dev.str));
 }

 void Uploader::uploadGainmap(struct deviceData stream)
 {
	 DEBUG("Gainmap upload ...");
	 HANDLE_CUDA_ERROR(hipSetDevice(stream.device));
	 DEBUG("hipMemcpy(" << stream.gain << ", " << stream.gain_host->at(0).data() << ", " << stream.gain_host->at(0).getSizeBytes() * 3 << ", hipMemcpyHostToDevice);");
	 HANDLE_CUDA_ERROR(hipMemcpy(stream.gain, stream.gain_host->at(0).data(), stream.gain_host->at(0).getSizeBytes() * 3, hipMemcpyHostToDevice));
	 DEBUG("Done!");
 }

 void Uploader::uploadPedestalmap(struct deviceData stream)
 {
	 DEBUG("Pedestalmap upload ...");
	 HANDLE_CUDA_ERROR(hipSetDevice(stream.device));
	 HANDLE_CUDA_ERROR(hipMemcpy(stream.pedestal, stream.pedestal_host->at(0).data(), stream.pedestal_host->at(0).getSizeBytes() * 3, hipMemcpyHostToDevice));
	 DEBUG("Done!");
 }

 void Uploader::downloadGainmap(struct deviceData stream)
 {
	 DEBUG("Gainmap upload ...");
	 HANDLE_CUDA_ERROR(hipSetDevice(stream.device));
	 DEBUG("hipMemcpy(" << stream.gain_host->at(0).data() << ", " << stream.gain << ", " << stream.gain_host->at(0).getSizeBytes() * 3 << ", hipMemcpyHostToDevice);");
	 HANDLE_CUDA_ERROR(hipMemcpy(stream.gain_host->at(0).data(), stream.gain, stream.gain_host->at(0).getSizeBytes() * 3, hipMemcpyDeviceToHost));
	 DEBUG("Done!");
 }

 void Uploader::downloadPedestalmap(struct deviceData stream)
 {
	 DEBUG("Pedestalmap doanload ...");
	 HANDLE_CUDA_ERROR(hipSetDevice(stream.device));
	 HANDLE_CUDA_ERROR(hipMemcpy(stream.pedestal_host->at(0).data(), stream.pedestal, stream.pedestal_host->at(0).getSizeBytes() * 3, hipMemcpyDeviceToHost));
	 DEBUG("Done!");
 }

 bool Uploader::calcFrames(std::vector<Datamap>& data)
 {
	 DEBUG("calcFrames");
	 std::vector<Photonmap> photonMaps;
	 photonMaps.reserve(GPU_FRAMES);

	 if(data.empty())
		 return false;

	 struct deviceData dev;
	 if(!resources.pop(dev))
		 return false;

	 uploadToGPU(dev, data);

	 //TODO: FIX KERNEL CALL
	 calculate<<</*DEVI.size() * NODES_PER_GPU / 128*/ 1, 128, 3 * (sizeof(uint16_t) + sizeof(double)) * 128, dev.str>>>(uint16_t(dimX * dimY / devices.size()), devices[i].pedestal, devices[i].gain, devices[i].data, uint16_t(GPU_FRAMES), devices[i].photons);
	 CHECK_CUDA_KERNEL;
	 downloadFromGPU(dev);

	 DEBUG("Creating callback ...");
	 HANDLE_CUDA_ERROR(hipStreamAddCallback(dev.str, Uploader::callback, &dev.id, 0));

	 DEBUG("End calcFrames");
	 return true;
 }

void Uploader::uploadToGPU(struct deviceData& dev, std::vector<Datamap>& data)
{
    HANDLE_CUDA_ERROR(hipSetDevice(dev.device));
    HANDLE_CUDA_ERROR(hipMemcpyAsync(dev.data, data.data(), data.size() * sizeof(data[0]), hipMemcpyHostToDevice, dev.str));
}

void Uploader::downloadFromGPU(struct deviceData& dev)
{
    DEBUG("Entering downloadFromGPU (str=" << dev.str << ")");
    std::size_t numPhotons = dimX * dimY * GPU_FRAMES;
    DEBUG("numPhotons = " << numPhotons);
    // TODO: find a better way than malloc
    uint16_t* photonData = (uint16_t*)malloc(numPhotons * sizeof(uint16_t));
    if (!photonData) {
        fputs("FATAL ERROR (Memory): Allocation failed!", stderr);
        exit(EXIT_FAILURE);
    }
    DEBUG(numPhotons * sizeof(uint16_t) << " Bytes allocated");
    DEBUG("hipMemcpyAsync(" << photonData << ", " << dev.photons << ", " << numPhotons * sizeof(uint16_t) << ", hipMemcpyDeviceToHost, " << dev.str << ");");

    HANDLE_CUDA_ERROR(hipSetDevice(dev.device));
    HANDLE_CUDA_ERROR(hipMemcpyAsync(photonData, dev.photons, numPhotons * sizeof(uint16_t), hipMemcpyDeviceToHost, dev.str));

    DEBUG("data downloaded");

    for (size_t i = 0; i < numPhotons; i += dimX * dimY) {
        dev.photon_host.emplace_back(dimX, dimY, &photonData[i]);
    }
    DEBUG("Data written. downloadFromGPU done!");
}

