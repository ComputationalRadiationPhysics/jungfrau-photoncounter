#include "upload.hpp"

static void handleCudaError(hipError_t error, const char* file, int line) {
	if(err != hipSuccess) {
		char errorString[1000];
		snprintf(errorString, 1000, "%s in %s at line %d!\n", hipGetErrorString(error), file, line);
		perror(errorString);
		exit(EXIT_FAILURE);
	}
}

Uploader::Uploader(Gainmap& gain, Pedestalmap& pedastel) : gain(gain), pedestal(pedestal), input_buffer(RINGBUFFER_SIZE), output_buffer(RINGBUFFER_SIZE), mem_usage(0){
	//TODO: handle pointer for multiple gpus here
	HANDLE_CUDA_ERROR(hipMalloc((void**)&gain_device, gain->getSizeBytes()));
	HANDLE_CUDA_ERROR(hipMalloc((void**)&pedestal_device, pedestal->getSizeBytes()));
	//TODO: find a better solution than this ugly hack below (using pedestal->getSizeBytes())!
	HANDLE_CUDA_ERROR(hipMalloc((void**)&data_device, pedestal->getSizeBytes()));
	//TODO: USE PROPER OUTPUTMAP TYPE!!!!!!!!!!
	HANDLE_CUDA_ERROR(hipMalloc((void**)&photons_device, pedestal->getSizeBytes() * 2));
	uploadGainmap(*gain);
	uploadPedestalmap(*pedestal);
	current_block.reserve(GPU_FRAMES);
}

Uploader::~Uploader() {
	HANDLE_CUDA_ERROR(hipFree(gain_device));
	HANDLE_CUDA_ERROR(hipFree(pedestal_device));
}

void Uploader::upload(std::vector<Datamap> data) {
	size_t i = 0;
	while(i < data.size()) {
		current_block.push_back(data[i]);
		if(current_block.size() == GPU_FRAMES) {
			input_buffer.push(current_block);
			current_block.clear();
		}
	}
	//TODO: use ringbuffer of threads???
}

std::vector<Datamap> Uploader::download() {
	std::vector<Datamap> ret;
	output_buffer.pop(ret);
	return ret;
}

void Uploader::uploadGainmap() {
	HANDLE_CUDA_ERROR(hipMemcpy(gain_device, gain->data(), gain->getSizeBytes(), hipMemcpyHostToDevice));
}

void Uploader::uploadPedestalmap() {
	HANDLE_CUDA_ERROR(hipMemcpy(pedestal_device, pedestal->data(), pedestal->getSizeBytes(), hipMemcpyHostToDevice));
}

void Uploader::downloadGainmap() {
	HANDLE_CUDA_ERROR(hipMemcpy(gain->data(), gain_device, gain->getSizeBytes(), hipMemcpyDeviceToHost));
}

void Uploader::downloadPedestalmap() {
	HANDLE_CUDA_ERROR(hipMemcpy(pedestal->data(), pedestal_device, pedestal->getSizeBytes(), hipMemcpyDeviceToHost));
}

void Uploader::calcFrames(std::vector<Datamap>& data) {
	hipStream_t str;
	//TODO: find something better than malloc here???
	float* photons = malloc();
	HANDLE_CUDA_ERROR(cudaCreateStream(&str));
	HANDLE_CUDA_ERROR(hipMemcpyAsync(data_device, data->data(), data->size() * sizeof(*data[0]), hipMemcpyHostToDevice, str));

	//TODO: use barrier or something similar here
	calculate<<<1/*blocks*/, 2/*threads*/, 0, str>>>(pedestal_device, gain_device, data_device, GPU_FRAMES, photons_device);

	//TODO: USE PROPER RETURN MAP TYPE!!!!
	HANDLE_CUDA_ERROR(hipMemcpy(photons_device, photons, data->size() * sizeof(photons_device[0])));
	for(size_t i = 0; i < data->size(); ++i) {
		//TODO: copy data to correct maptype
	}
}

/*
void Uploader::uploadToGPU() {
	std::vector<Datamap> to_upload;
	hipStream_t str;
	input_buffer.pop(to_upload);
	//TODO: better error handling!
	if(to_upload.empty())
		return;
	//TODO: figure out how to multithread this properly!
	HANDLE_CUDA_ERROR(hipStreamCreate(&str));
	streams.push_back(str);
	hipMemcpyAsync(data_device, to_upload.data(), to_upload.size() * sizeof(to_upload.data()[0]), hipMemcpyHostToDevice, str);
}

void Uploader::downloadFromGPU() {
	//TODO: figure out how to multithread this properly!
}
*/
