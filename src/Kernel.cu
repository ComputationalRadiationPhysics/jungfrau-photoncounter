#include "hip/hip_runtime.h"
#include "Kernel.hpp"

__global__ void calculate(uint16_t mapsize, uint16_t* pede, double* gain,
                          uint16_t* data, uint16_t num, uint16_t* photon)
{
    extern __shared__ uint16_t sPede[];
    extern __shared__ uint16_t sGain[];

    uint16_t id = blockIdx.x * blockDim.x + threadIdx.x;

    sPede[id] = pede[id];
    sPede[mapsize + id] = pede[mapsize + id];
    sPede[(mapsize * 2) + id] = pede[(mapsize * 2) + id];
    sGain[id] = gain[id];
    sGain[mapsize + id] = gain[mapsize + id];
    sGain[(mapsize * 2) + id] = gain[(mapsize * 2) + id];

    __syncthreads();

    for (int i = 0; i < num; i++) {
        uint16_t dataword = data[(mapsize * i) + id];
        float energy;

        switch ((dataword & 0xc000) >> 14) {
        case 0:
            energy =
                (dataword & 0x3fff - sPede[id]) * sGain[id];
            break;
        case 1:
            energy =
                (sPede[mapsize + id] - dataword & 0x3fff) * sGain[mapsize + id];
            break;
        case 3:
            energy =
                (sPede[(2 * mapsize) + id] - dataword & 0x3fff) *
                sGain[(2 * mapsize) + id];
            break;
        default:
            energy = 0;
            break;
        }

        photon[(mapsize * i) + id] = int((energy + 6.2) / 12.4);
    }
}

__global__ void calibrate(uint16_t mapsize, uint16_t* data, uint16_t num,
                          uint16_t* pede)
{
    uint16_t id = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i < 1000; i++) {
        if (i == 0) {
            pede[id] = data[(mapsize * i) + id] & 0x3fff;
        }
        else {
            pede[id] += data[(mapsize * i) + id] & 0x3fff;
        }
    }

    for (int i = 1000; i < num; i++) {
        pede[id] =
            (pede[id] + data[(mapsize * i) + id] & 0x3fff) - (pede[id] / i);
    }

    pede[id] = round((double)pede[id] / 1000);
}
