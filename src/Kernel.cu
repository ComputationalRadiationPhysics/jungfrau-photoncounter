#include "hip/hip_runtime.h"
#include "Kernel.hpp"

__global__ void calculate(uint16_t* pede, double* gain, uint16_t* data, uint16_t num, float* energy) {
    const uint16_t arraysize = sizeof(pede) / sizeof(pede[0]);
    const uint16_t mapsize = arraysize / 3;
    __shared__ uint16_t sPede[arraysize];
    __shared__ uint16_t sGain[arraysize];

    uint16_t id = blockIdx.x * blockDim.x + threadIdx.x;

    sPede[id] = pede[id];
    sGain[id] = gain[id];

    __syncthreads();

    for(int i = 0; i < num; i++) {
        uint16_t dataword = data[(mapsize*i)+id];

        switch((dataword&0xc000) >> 14) {
            case 0: energy[(mapsize*i)+id] = 
                    (dataword&0x3fff - sPede[id]) * sGain[id];
                    break;
            case 1: energy[(mapsize*i)+id] =
                    (sPede[mapsize+id] - dataword&0x3fff) * 
                    sGain[mapsize*id];
                    break;
            case 3: energy[(mapsize*i)+id] =
                    (sPede[(2*mapsize)+id] - dataword&0x3fff) *
                    sGain[(2*mapsize)+id];
                    break;
            default: 
                    energy[(mapsize*i)+id] = 0;
                    break;
        }
    } 
}

__global__ void calibrate(uint16_t* data, uint16_t num, uint16_t* pede) {
    const uint16_t mapsize = sizeof(pede) / sizeof(pede[0]);

    uint16_t id = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = 0; i < 1000; i++) {
        if(i == 0) {
            pede[id] = data[(mapsize*i)+id]&0x3fff;
        } else {
            pede[id] += data[(mapsize*i)+id]&0x3fff;
        }
    }

    for(int i = 1000; i < num; i++) {
        pede[id] = (pede[id] + data[(mapsize*i)+id]&0x3fff) - (pede[id] / i);      
    }

    pede[id] = round((double)pede[id] / 1000);
}

//delete this, it's just so it compiles with nvcc
int main() {
    return 0;
}
