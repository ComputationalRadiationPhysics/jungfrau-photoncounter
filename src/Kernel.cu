#include "hip/hip_runtime.h"
#include "Kernel.hpp"

__global__ void calculate(uint32_t mapsize, uint64_t* pede, double* gain,
                          uint16_t* data, uint32_t num, uint16_t* photon)
{
    // locally save gain/ped values for the associated pixel
    uint16_t lPede[3];
    uint16_t lMovAvg;
    uint32_t lCounter;
    uint16_t lGain[3];

    // find id and copy gain/pede maps
    uint32_t id = blockIdx.x * blockDim.x + threadIdx.x;

    lPede[0] = pede[id] & 0x000000000000ffff;
    lPede[1] = pede[mapsize + id] & 0x000000000000ffff;
    lPede[2] = pede[(2 * mapsize) + id] & 0x000000000000ffff;
    lMovAvg = (pede[id] & 0x00000000ffff0000) >> 16;
    lCounter = (pede[id] & 0xffffffff00000000) >> 32;
    lGain[0] = gain[id];
    lGain[1] = gain[mapsize + id];
    lGain[2] = gain[(mapsize * 2) + id];

    // calc the energy value for one pixel in each frame
    for (int i = 0; i < num; ++i) {
        // 8*(i++) is the header of each frame
        uint16_t dataword = data[(mapsize * i) + id + (8 * (i + 1))];
        uint16_t adc = dataword & 0x3fff;
        float energy;

        switch ((dataword & 0xc000) >> 14) {
        case 0:
            if (adc < 100) {
                // calibration for dark pixels
                lMovAvg = lMovAvg + adc - (lMovAvg / lCounter);
                if (lCounter < 4294000000)
                    lCounter++;

                lPede[0] = lMovAvg / lCounter;
            }
            else
                energy = (adc - lPede[0]) * lGain[0];
            break;
        case 1:
            energy = (lPede[1] - adc) * lGain[1];
            break;
        case 3:
            energy = (lPede[2] - adc) * lGain[2];
            break;
        default:
            energy = 0;
            break;
        }
        photon[(mapsize * i) + id + (8 * (i + 1))] = int((energy + 6.2) / 12.4);

        // copy the header
        if (threadIdx.x < 8) {
            photon[(mapsize * i) + (threadIdx.x * (i + 1))] =
                data[(mapsize * i) + (threadIdx.x * (i + 1))];
        }
    }

    // save new pedestal value
    pede[id] = ((uint64_t)lCounter << 32) | ((uint64_t)lMovAvg << 16) |
               (uint64_t)lPede[0];
}

__global__ void calibrate(uint32_t mapsize, uint32_t num, uint32_t currentnum,
                          uint16_t* data, uint64_t* pede)
{
    uint16_t id = blockIdx.x * blockDim.x + threadIdx.x;

    // 32 bit counter; 16 bit moving average; 16 bit offset
    // for calibration only average = offset
    uint32_t counter;
    uint16_t average;

    if (currentnum == 0) {
        counter = 0;
        average = 0;
    } else {
        counter = pede[id] & 0xffffffff00000000;
        average = pede[id] & 0x00000000ffff0000;
    }

    // base value for pedestal stage 0
    for (int i = currentnum; i < 1000 && i < num; i++) {
        average += data[(mapsize * i) + id] & 0x3fff;
        counter++;
    }

    average = round((double)average / counter);

    // combine all values into one 64 bit dataword, so we only need one map
    pede[id] = ((uint64_t)counter << 32) | ((uint64_t)average << 16) |
               (uint64_t)average;

    // base value for pedestal stage 1
    average = 0;
    counter = 0;
    for (int i = currentnum; i > 999 && i < 2000 && i < (999 + num); i++) {
        average += data[(mapsize * i) + id] & 0x3fff;
        counter++;
    }

    average = round((double)average / counter);

    pede[mapsize + id] = ((uint64_t)counter << 32) | ((uint64_t)average << 16) |
                         (uint64_t)average;

    // base value for pedestal stage 3
    average = 0;
    counter = 0;
    for (int i = currentnum; i > 1999 && i < 2999 + i < (1999 + num); i++) {
        average += data[(mapsize * i) + id] & 0x3fff;
        counter++;
    }

    average = round((double)average / counter);

    pede[(mapsize * 2) + id] = ((uint64_t)counter << 32) |
                               ((uint64_t)average << 16) | (uint64_t)average;
}
