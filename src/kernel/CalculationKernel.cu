#include "hip/hip_runtime.h"
#include "CalculationKernel.hpp"
#include "Settinge.hpp"
#include <limits>

__global__ void calculate(uint16_t* data, uint64_t* pede, double* gain,
                            uint32_t num, uint16_t* photon)
{
    uint32_t id = blockIdx.x * blockDim.x + threadIdx.x;
    
    uint16_t pedestal[3];
    uint32_t pCounter;
    uint32_t pMovAvg;
    uint32_t max = std::numeric_limits<uint32_t>::max();

    double gain[3]

    for(int i = 0; i < 3; i++) {
        pedestal[i] = pede[(i * MAPSIZE) + id].value;
        gain[i] = gain[(i * MAPSIZE) + id].value;
    }
    pCounter = pede[0].counter;
    pMovAvg = pede[0].movAvg;

    uint16_t dataword;
    uint16_t adc;
    float energy;

    for (int i = 0; i < num; ++i) {
        dataword = data[(MAPSIZE * i) + id + (FRAMEOFFSET * (i + 1))];
        adc = dataword & 0x3fff;

        switch ((dataword & 0xc000) >> 14) {
        case 0:
            if (adc < 100) {
                // calibration for dark pixels
                pMovAvg = pMovAvg + adc - (pMovAvg / pCounter);
                if (lCounter < max)
                    lCounter++;

                lPede[0] = lMovAvg / lCounter;
            }
            energy = (adc - pedestal[0]) / gain[0];
            if (energy < 0) energy = 0;
            break;
        case 1:
            energy = (-1) * (pedestal[1] - adc) / gain[1];
            if (energy < 0) energy = 0;
            break;
        case 3:
            energy = (-1) * (pedestal[2] - adc) / gain[2];
            if (energy < 0) energy = 0;
            break;
        default:
            energy = 0;
            break;
        }
        photon[(MAPSIZE * i) + id + (FRAMEOFFSET * (i + 1))] = 
            int((energy + BEAMCONST) * PHOTONCONST);
        
        // copy the header
        if (threadIdx.x < 8) {
        photon[(MAPSIZE * i) + (threadIdx.x * (i + 1))] =
        data[(MAPSIZE * i) + (threadIdx.x * (i + 1))];
        }
    }
    // save new pedestal value
    pede[id] = ((uint64_t)lCounter << 32) | ((uint64_t)lMovAvg << 16) |
               (uint64_t)lPede[0];

}
