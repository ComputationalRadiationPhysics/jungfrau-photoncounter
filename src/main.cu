#include "Filecache.hpp"
#include "Upload.hpp"
#include "Bitmap.hpp"
#include <iomanip>
#include <iostream>
#include <string>

const std::size_t NUM_UPLOADS = 2;

int main()
{
    DEBUG("Entering main ...");
    Filecache fc(1024UL * 1024 * 1024 * 16);
    std::vector<Pedestalmap> pedestal =
        fc.loadMaps<Pedestalmap>("data_pool/px_101016/pedeMaps.bin", 1024, 512);
    DEBUG("Pedestalmap loaded!");
    std::vector<Datamap> data = fc.loadMaps<Datamap>("data_pool/px_101016/Insu_6_tr_1_45d_250us__B_000000.dat", 1024, 512);
    DEBUG("Datamap loaded!");
    std::vector<Gainmap> gain = fc.loadMaps<Gainmap>(
        "data_pool/px_101016/gainMaps_M022.bin", 1024, 512);
    DEBUG("Gainmap loaded!");

    std::array<Pedestalmap, 3> pedestal_array = {pedestal[0], pedestal[1], pedestal[2]};
    std::array<Gainmap, 3> gain_array = {gain[0], gain[1], gain[2]};



	//TODO: remove below; this is only used because the loaded pedestal maps semm to be incorrect
	//force pedestal to 0
	uint16_t* p = pedestal.at(0).data();
	for(std::size_t i = 0; i < pedestal.at(0).getSizeBytes() * 3; ++i){
		p[i] = 0;
	}

	Datamap dtest = data.at(0);
	Bitmap::Image img(1024, 512);
	for(int j = 0; j < 1024; j++) {
		for(int k=0; k < 512; k++) {
			int h = dtest(j, k) / 256;
			Bitmap::Rgb color = {(unsigned char)h, (unsigned char)h, (unsigned char)h};
			img(j, k) = color;
		}
	}
	img.writeToFile("dtest.bmp");

	Pedestalmap ptest = pedestal.at(0);
	Bitmap::Image img2(1024, 512);
	for(int j = 0; j < 1024; j++) {
		for(int k=0; k < 512; k++) {
			int h = ptest(j, k) / 256;
			Bitmap::Rgb color = {(unsigned char)h, (unsigned char)h, (unsigned char)h};
			img2(j, k) = color;
		}
	}
	img2.writeToFile("ptest.bmp");

	Gainmap gtest = gain.at(0);
	Bitmap::Image img3(1024, 512);
	for(int j = 0; j < 1024; j++) {
		for(int k=0; k < 512; k++) {
			int h = gtest(j, k) * 200;
			Bitmap::Rgb color = {(unsigned char)h, (unsigned char)h, (unsigned char)h};
			img3(j, k) = color;
		}
	}
	img3.writeToFile("gtest.bmp");


	int num = 0;
	HANDLE_CUDA_ERROR(hipGetDeviceCount(&num));

    Uploader up(gain_array, pedestal_array, 1024, 512, num);
    DEBUG("Uploader created!");

	std::vector<Datamap> data_backup(data);
	std::vector<Photonmap> ready;
	ready.reserve(GPU_FRAMES);

	DEBUG("starting upload!");

    int bitteFunktioniere = 1;
	for(std::size_t i = 1; i <= NUM_UPLOADS; ++i) {
		while(!up.upload(data) && !data.empty()) {
			while(!(ready = up.download()).empty()) {

                if (bitteFunktioniere == 1) {
					Photonmap test = ready.at(0);
                    Bitmap::Image img(1024, 512);
                    for(int j = 0; j < 1024; j++) {
                        for(int k=0; k < 512; k++) {
                            int h = test(j, k) / 256;
                            Bitmap::Rgb color = {(unsigned char)h, (unsigned char)h, (unsigned char)h};
                            img(j, k) = color;
                        }
                    }
                    img.writeToFile("test.bmp");


					test = ready.at(1);
                    for(int j = 0; j < 1024; j++) {
                        for(int k=0; k < 512; k++) {
                            int h = test(j, k) / 256;
                            Bitmap::Rgb color = {(unsigned char)h, (unsigned char)h, (unsigned char)h};
                            img(j, k) = color;
                        }
                    }
                    img.writeToFile("test1.bmp");

					test = ready.at(2);
                    for(int j = 0; j < 1024; j++) {
                        for(int k=0; k < 512; k++) {
                            int h = test(j, k) / 256;
                            Bitmap::Rgb color = {(unsigned char)h, (unsigned char)h, (unsigned char)h};
                            img(j, k) = color;
                        }
                    }
                    img.writeToFile("test2.bmp");

					test = ready.at(3);
                    for(int j = 0; j < 1024; j++) {
                        for(int k=0; k < 512; k++) {
                            int h = test(j, k) / 256;
                            Bitmap::Rgb color = {(unsigned char)h, (unsigned char)h, (unsigned char)h};
                            img(j, k) = color;
                        }
                    }
                    img.writeToFile("test3.bmp");



                    bitteFunktioniere = 0;
                }
				free(ready[0].data());
				DEBUG("freeing in main");
			}
		}
		data = data_backup;
		DEBUG("Uploaded " << i << "/" << NUM_UPLOADS);
	}

	up.synchronize();

    return 0;
}
